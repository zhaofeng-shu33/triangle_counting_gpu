#include "hip/hip_runtime.h"
// Copyright 2019 zhaofeng-shu33
#include "counting_gpu.h"

#include "gpu-thrust.h"
#include "timer.h"

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <iostream>
#include <vector>

using namespace std;

#define NUM_THREADS 64
#define NUM_BLOCKS_GENERIC 112
#define NUM_BLOCKS_PER_MP 8

template<bool ZIPPED>
__global__ void CalculateNodePointers(int n, int m, int* edges, int* nodes) {
  int from = blockDim.x * blockIdx.x + threadIdx.x;
  int step = gridDim.x * blockDim.x;
  for (int i = from; i <= m; i += step) {
    int prev = i > 0 ? edges[ZIPPED ? (2 * (i - 1) + 1) : (m + i - 1)] : -1;
    int next = i < m ? edges[ZIPPED ? (2 * i + 1) : (m + i)] : n;
    for (int j = prev + 1; j <= next; ++j)
      nodes[j] = i;
  }
}

__global__ void UnzipEdges(int m, int* edges, int* unzipped_edges) {
  int from = blockDim.x * blockIdx.x + threadIdx.x;
  int step = gridDim.x * blockDim.x;
  for (int i = from; i < m; i += step) {
    unzipped_edges[i] = edges[2 * i];
    unzipped_edges[m + i] = edges[2 * i + 1];
  }
}

__global__ void CalculateTriangles(
    int m, const int* __restrict__ edges, const int* __restrict__ nodes,
    uint64_t* results, int deviceCount = 1, int deviceIdx = 0) {
  int from =
    gridDim.x * blockDim.x * deviceIdx +
    blockDim.x * blockIdx.x +
    threadIdx.x;
  int step = deviceCount * gridDim.x * blockDim.x;
  uint64_t count = 0;

  for (int i = from; i < m; i += step) {
    int u = edges[i], v = edges[m + i];

    int u_it = nodes[u], u_end = nodes[u + 1];
    int v_it = nodes[v], v_end = nodes[v + 1];

    int a = edges[u_it], b = edges[v_it];
    while (u_it < u_end && v_it < v_end) {
      int d = a - b;
      if (d <= 0)
        a = edges[++u_it];
      if (d >= 0)
        b = edges[++v_it];
      if (d == 0)
        ++count;
    }
  }

  results[blockDim.x * blockIdx.x + threadIdx.x] = count;
}

__global__ void CalculateTriangleSplit(int edge_len,
    const int* __restrict__ edges, const int* __restrict__ edges_i, const int* __restrict__ edges_j, const uint64_t* __restrict__ nodes,
    uint64_t* results, const uint64_t* __restrict__ dev_node_index, int i, int j) {
  int from =
    blockDim.x * blockIdx.x +
    threadIdx.x;
  int step = gridDim.x * blockDim.x;
  uint64_t count = 0;
  // itering over edges_i
  for (uint64_t r = from; r < edge_len; r += step) {
    int u = edges[r], v = edges[r + edge_len];
    if(nodes[u] >= dev_node_index[i+1] || nodes[u + 1] < dev_node_index[i] || nodes[v] >= dev_node_index[j + 1] || nodes[v + 1] < dev_node_index[j])
        continue;
    uint64_t u_it = nodes[u] - dev_node_index[i];
    uint64_t u_end = nodes[u + 1] - dev_node_index[i];
    uint64_t v_it = nodes[v] - dev_node_index[j];
    uint64_t v_end = nodes[v + 1] - dev_node_index[j];
    // if u_it or v_it not in edges, continue the loop
    int a = edges_i[u_it], b = edges_j[v_it];
    while (u_it < u_end && v_it < v_end) {
      int d = a - b;
      if (d <= 0)
        a = edges_i[++u_it];
      if (d >= 0)
        b = edges_j[++v_it];
      if (d == 0)
        ++count;
    }
  }

  results[blockDim.x * blockIdx.x + threadIdx.x] = count;
}
void CudaAssert(hipError_t status, const char* code, const char* file, int l) {
  if (status == hipSuccess) return;
  cerr << "Cuda error: " << code << ", file " << file << ", line " << l << endl;
  exit(1);
}

#define CUCHECK(x) CudaAssert(x, #x, __FILE__, __LINE__)

int NumberOfMPs() {
  int dev, val;
  CUCHECK(hipGetDevice(&dev));
  CUCHECK(hipDeviceGetAttribute(&val, hipDeviceAttributeMultiprocessorCount, dev));
  return val;
}

size_t GlobalMemory() {
  int dev;
  hipDeviceProp_t prop;
  CUCHECK(hipGetDevice(&dev));
  CUCHECK(hipGetDeviceProperties(&prop, dev));
  return prop.totalGlobalMem;
}

uint64_t GpuForward(int* edges, int num_nodes, uint64_t num_edges) {
  return MultiGpuForward(edges, 1, num_nodes, num_edges);
}

//! get the split_num based of edge num and node num
int GetSplitNum(int num_nodes, uint64_t num_edges) {
    uint64_t mem = (uint64_t)GlobalMemory(); // in Byte
    mem -= (uint64_t)num_nodes * 8; // uint64_t
    return (1 + 16 * num_edges / mem);
}

uint64_t GpuForwardSplit(int* edges, int num_nodes, uint64_t num_edges, int split_num) {
#if TIMECOUNTING
  Timer* timer = Timer::NewTimer();
#endif
  CUCHECK(hipSetDevice(0));
  const int NUM_BLOCKS = NUM_BLOCKS_PER_MP * NumberOfMPs();

  uint64_t m = num_edges;
  int n = num_nodes;

  int *dev_edges, *dev_edges_i, *dev_edges_j;
  uint64_t* dev_nodes;
  uint64_t* host_nodes;
  swap_array(edges, num_edges);
  CUCHECK(hipMalloc(&dev_nodes, (n + 1) * sizeof(uint64_t)));
  // compute node pointers in CPU
  host_nodes = new uint64_t [n + 1];
  // Calculate NodePointers
  for (uint64_t i = 0; i <= m; i++) {
     int prev = i > 0 ? edges[m + i - 1] : -1;
     int next = i < m ? edges[m + i] : n;
     for (int j = prev + 1; j <= next; j++)
       host_nodes[j] = i;  
  }
  // copy node pointers from CPU memory to GPU memory
  CUCHECK(hipMemcpyAsync(
      dev_nodes, host_nodes, (n + 1) * sizeof(uint64_t), hipMemcpyHostToDevice));
  CUCHECK(hipDeviceSynchronize());

  uint64_t result = 0;
   
  // calculate split index in host_nodes which makes the split even
  uint64_t* node_index = new uint64_t[split_num + 1];
  uint64_t max_len = get_split(host_nodes, n + 1, split_num, node_index);
  CUCHECK(hipMalloc(&dev_edges, 2 * sizeof(int) * max_len));
  CUCHECK(hipMalloc(&dev_edges_i, sizeof(int) * max_len));
  CUCHECK(hipMalloc(&dev_edges_j, sizeof(int) * max_len));
  uint64_t* dev_results;
  CUCHECK(hipMalloc(&dev_results,
	  NUM_BLOCKS * NUM_THREADS * sizeof(uint64_t)));
  hipFuncSetCacheConfig(reinterpret_cast<const void*>(CalculateTriangleSplit), hipFuncCachePreferL1);

  uint64_t* dev_node_index;
  CUCHECK(hipMalloc(&dev_node_index, (split_num + 1) * sizeof(uint64_t)));
  CUCHECK(hipMemcpy(dev_node_index, node_index, (split_num + 1)* sizeof(uint64_t), hipMemcpyHostToDevice));
  for(int t = 0; t < split_num; t++)
    for(int i = 0; i < split_num; i++)
      for(int j = 0; j < split_num; j++){
          uint64_t data_offset = node_index[t + 1] - node_index[t];
          CUCHECK(hipMemcpy(dev_edges, edges + node_index[t], sizeof(int) * data_offset, hipMemcpyHostToDevice));
          CUCHECK(hipMemcpy(dev_edges + data_offset, edges + m + node_index[t], sizeof(int) * data_offset,
           hipMemcpyHostToDevice));
          CUCHECK(hipMemcpy(dev_edges_i, edges + node_index[i], sizeof(int) * (node_index[i + 1] - node_index[i]),
            hipMemcpyHostToDevice));
          CUCHECK(hipMemcpy(dev_edges_j, edges + node_index[j], sizeof(int) * (node_index[j + 1] - node_index[j]),
            hipMemcpyHostToDevice));
          // node id dev_node_index[i]~dev_node_index[i+1] and dev_node_index[j]~dev_node_index[j+1]
 	    CalculateTriangleSplit<<<NUM_BLOCKS, NUM_THREADS>>>(
		data_offset, dev_edges, dev_edges_i, dev_edges_j, dev_nodes, dev_results, dev_node_index, i, j);
	    CUCHECK(hipDeviceSynchronize());
	    // Reduce
	  result += SumResults(NUM_BLOCKS * NUM_THREADS, dev_results);
    }
#if TIMECOUNTING    
  timer->Done("Calculate triangles used time: ");
#endif
  CUCHECK(hipFree(dev_results));
  CUCHECK(hipFree(dev_edges));
  CUCHECK(hipFree(dev_edges_i));
  CUCHECK(hipFree(dev_edges_j));
  CUCHECK(hipFree(dev_nodes));
#if TIMECOUNTING
  delete timer;
#endif
  delete node_index;
  free(host_nodes);
  return result;
}

uint64_t MultiGpuForward(int* edges, int device_count, int num_nodes, uint64_t num_edges) {
#if TIMECOUNTING
  Timer* timer = Timer::NewTimer();
#endif
  CUCHECK(hipSetDevice(0));
  const int NUM_BLOCKS = NUM_BLOCKS_PER_MP * NumberOfMPs();

  uint64_t m = num_edges;
  int n = num_nodes;

  int* dev_edges;
  int* dev_nodes;

  
  int* dev_temp;
  CUCHECK(hipMalloc(&dev_temp, m * 2 * sizeof(int)));
  CUCHECK(hipMemcpyAsync(
      dev_temp, edges, m * 2 * sizeof(int), hipMemcpyHostToDevice));
  CUCHECK(hipDeviceSynchronize());
  // Memcpy edges from host to device
  SortEdges(m, dev_temp);
  CUCHECK(hipDeviceSynchronize());
  // Sort edges

  CUCHECK(hipMalloc(&dev_edges, m * 2 * sizeof(int)));
  UnzipEdges<<<NUM_BLOCKS, NUM_THREADS>>>(m, dev_temp, dev_edges);
  CUCHECK(hipFree(dev_temp));
  CUCHECK(hipDeviceSynchronize());
  // Unzip edges


  CUCHECK(hipMalloc(&dev_nodes, (n + 1) * sizeof(int)));
  CalculateNodePointers<false><<<NUM_BLOCKS, NUM_THREADS>>>(
      n, m, dev_edges, dev_nodes);
  CUCHECK(hipDeviceSynchronize());
  // Calculate nodes array for one-way unzipped edges
  uint64_t result = 0;

  
  uint64_t* dev_results;
  CUCHECK(hipMalloc(&dev_results,
        NUM_BLOCKS * NUM_THREADS * sizeof(uint64_t)));
  hipFuncSetCacheConfig(reinterpret_cast<const void*>(CalculateTriangles), hipFuncCachePreferL1);
  hipProfilerStart();
  CalculateTriangles<<<NUM_BLOCKS, NUM_THREADS>>>(
      m, dev_edges, dev_nodes, dev_results);
  CUCHECK(hipDeviceSynchronize());
  hipProfilerStop();
  // Reduce
  result = SumResults(NUM_BLOCKS * NUM_THREADS, dev_results);
#if TIMECOUNTING    
  timer->Done("Calculate triangles used time: ");
#endif
  CUCHECK(hipFree(dev_results));
  

  CUCHECK(hipFree(dev_edges));
  CUCHECK(hipFree(dev_nodes));
#if TIMECOUNTING
  delete timer;
#endif
  return result;
}

void PreInitGpuContext(int device) {
  CUCHECK(hipSetDevice(device));
  CUCHECK(hipFree(NULL));
}
