#include "hip/hip_runtime.h"
#include "gpu.h"

#include "gpu-thrust.h"
#include "timer.h"

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <iostream>
#include <vector>
using namespace std;

#define NUM_THREADS 64
#define NUM_BLOCKS_GENERIC 112
#define NUM_BLOCKS_PER_MP 8

template<bool ZIPPED>
__global__ void CalculateNodePointers(int n, int m, int* edges, int* nodes) {
  int from = blockDim.x * blockIdx.x + threadIdx.x;
  int step = gridDim.x * blockDim.x;
  for (int i = from; i <= m; i += step) {
    int prev = i > 0 ? edges[ZIPPED ? (2 * (i - 1) + 1) : (m + i - 1)] : -1;
    int next = i < m ? edges[ZIPPED ? (2 * i + 1) : (m + i)] : n;
    for (int j = prev + 1; j <= next; ++j)
      nodes[j] = i;
  }
}

__global__ void CalculateFlags(int m, int* edges, int* nodes, bool* flags) {
  int from = blockDim.x * blockIdx.x + threadIdx.x;
  int step = gridDim.x * blockDim.x;
  for (int i = from; i < m; i += step) {
    int a = edges[2 * i];
    int b = edges[2 * i + 1];
    int deg_a = nodes[a + 1] - nodes[a];
    int deg_b = nodes[b + 1] - nodes[b];
    flags[i] = (deg_a < deg_b) || (deg_a == deg_b && a < b);
  }
}

__global__ void UnzipEdges(int m, int* edges, int* unzipped_edges) {
  int from = blockDim.x * blockIdx.x + threadIdx.x;
  int step = gridDim.x * blockDim.x;
  for (int i = from; i < m; i += step) {
    unzipped_edges[i] = edges[2 * i];
    unzipped_edges[m + i] = edges[2 * i + 1];
  }
}

__global__ void CalculateTriangles(
    int m, const int* __restrict__ edges, const int* __restrict__ nodes,
    uint64_t* results, int deviceCount = 1, int deviceIdx = 0) {
  int from =
    gridDim.x * blockDim.x * deviceIdx +
    blockDim.x * blockIdx.x +
    threadIdx.x;
  int step = deviceCount * gridDim.x * blockDim.x;
  uint64_t count = 0;

  for (int i = from; i < m; i += step) {
    int u = edges[i], v = edges[m + i];

    int u_it = nodes[u], u_end = nodes[u + 1];
    int v_it = nodes[v], v_end = nodes[v + 1];

    int a = edges[u_it], b = edges[v_it];
    while (u_it < u_end && v_it < v_end) {
      int d = a - b;
      if (d <= 0)
        a = edges[++u_it];
      if (d >= 0)
        b = edges[++v_it];
      if (d == 0)
        ++count;
    }
  }

  results[blockDim.x * blockIdx.x + threadIdx.x] = count;
}

void CudaAssert(hipError_t status, const char* code, const char* file, int l) {
  if (status == hipSuccess) return;
  cerr << "Cuda error: " << code << ", file " << file << ", line " << l << endl;
  exit(1);
}

#define CUCHECK(x) CudaAssert(x, #x, __FILE__, __LINE__)

int NumberOfMPs() {
  int dev, val;
  CUCHECK(hipGetDevice(&dev));
  CUCHECK(hipDeviceGetAttribute(&val, hipDeviceAttributeMultiprocessorCount, dev));
  return val;
}

size_t GlobalMemory() {
  int dev;
  hipDeviceProp_t prop;
  CUCHECK(hipGetDevice(&dev));
  CUCHECK(hipGetDeviceProperties(&prop, dev));
  return prop.totalGlobalMem;
}

Edges RemoveBackwardEdgesCPU(const Edges& unordered_edges) {
  int n = NumVertices(unordered_edges);
  int m = unordered_edges.size();

  vector<int> deg(n);
  for (int i = 0; i < m; ++i)
    ++deg[unordered_edges[i].first];

  vector< pair<int, int> > edges;
  edges.reserve(m / 2);
  for (int i = 0; i < m; ++i) {
    int s = unordered_edges[i].first, t = unordered_edges[i].second;
    if (deg[s] > deg[t] || (deg[s] == deg[t] && s > t))
      edges.push_back(make_pair(s, t));
  }

  return edges;
}

uint64_t MultiGPUCalculateTriangles(
    int n, int m, int* dev_edges, int* dev_nodes, int device_count) {
  vector<int*> multi_dev_edges(device_count);
  vector<int*> multi_dev_nodes(device_count);

  multi_dev_edges[0] = dev_edges;
  multi_dev_nodes[0] = dev_nodes;

  for (int i = 1; i < device_count; ++i) {
    CUCHECK(hipSetDevice(i));
    CUCHECK(hipMalloc(&multi_dev_edges[i], m * 2 * sizeof(int)));
    CUCHECK(hipMalloc(&multi_dev_nodes[i], (n + 1) * sizeof(int)));
    int dst = i, src = (i + 1) >> 2;
    CUCHECK(hipMemcpyPeer(
          multi_dev_edges[dst], dst, multi_dev_edges[src], src,
          m * 2 * sizeof(int)));
    CUCHECK(hipMemcpyPeer(
          multi_dev_nodes[dst], dst, multi_dev_nodes[src], src,
          (n + 1) * sizeof(int)));
  }

  vector<int> NUM_BLOCKS(device_count);
  vector<uint64_t*> multi_dev_results(device_count);

  for (int i = 0; i < device_count; ++i) {
    CUCHECK(hipSetDevice(i));
    NUM_BLOCKS[i] = NUM_BLOCKS_PER_MP * NumberOfMPs();
    CUCHECK(hipMalloc(
          &multi_dev_results[i],
          NUM_BLOCKS[i] * NUM_THREADS * sizeof(uint64_t)));
  }

  for (int i = 0; i < device_count; ++i) {
    CUCHECK(hipSetDevice(i));
    CUCHECK(hipFuncSetCacheConfig(reinterpret_cast<const void*>(CalculateTriangles), hipFuncCachePreferL1));
    CalculateTriangles<<<NUM_BLOCKS[i], NUM_THREADS>>>(
        m, multi_dev_edges[i], multi_dev_nodes[i], multi_dev_results[i],
        device_count, i);
  }

  uint64_t result = 0;

  for (int i = 0; i < device_count; ++i) {
    CUCHECK(hipSetDevice(i));
    CUCHECK(hipDeviceSynchronize());
    result += SumResults(NUM_BLOCKS[i] * NUM_THREADS, multi_dev_results[i]);
  }

  for (int i = 1; i < device_count; ++i) {
    CUCHECK(hipSetDevice(i));
    CUCHECK(hipFree(multi_dev_edges[i]));
    CUCHECK(hipFree(multi_dev_nodes[i]));
  }

  for (int i = 0; i < device_count; ++i) {
    CUCHECK(hipSetDevice(i));
    CUCHECK(hipFree(multi_dev_results[i]));
  }

  hipSetDevice(0);
  return result;
}

uint64_t GpuForward(const Edges& edges) {
  return MultiGpuForward(edges, 1);
}

uint64_t MultiGpuForward(const Edges& edges, int device_count) {
#if TIMECOUNTING
  Timer* timer = Timer::NewTimer();
#endif
  CUCHECK(hipSetDevice(0));
  const int NUM_BLOCKS = NUM_BLOCKS_PER_MP * NumberOfMPs();

  int m = edges.size(), n;

  int* dev_edges;
  int* dev_nodes;

  Edges& fwd_edges = edges;
  
  int* dev_temp;
  CUCHECK(hipMalloc(&dev_temp, m * 2 * sizeof(int)));
  CUCHECK(hipMemcpyAsync(
      dev_temp, fwd_edges.data(), m * 2 * sizeof(int), hipMemcpyHostToDevice));
  CUCHECK(hipDeviceSynchronize());
#if TIMECOUNTING
  timer->Done("Memcpy edges from host do device");
#endif 
  SortEdges(m, dev_temp);
  CUCHECK(hipDeviceSynchronize());
#if TIMECOUNTING
  timer->Done("Sort edges");
#endif
  CUCHECK(hipMalloc(&dev_edges, m * 2 * sizeof(int)));
  UnzipEdges<<<NUM_BLOCKS, NUM_THREADS>>>(m, dev_temp, dev_edges);
  CUCHECK(hipFree(dev_temp));
  CUCHECK(hipDeviceSynchronize());
#if TIMECOUNTING  
  timer->Done("Unzip edges");
#endif
  n = NumVerticesGPU(m, dev_edges);
  CUCHECK(hipMalloc(&dev_nodes, (n + 1) * sizeof(int)));
#if TIMECOUNTING  
  timer->Done("Calculate number of vertices");
#endif

  CalculateNodePointers<false><<<NUM_BLOCKS, NUM_THREADS>>>(
      n, m, dev_edges, dev_nodes);
  CUCHECK(hipDeviceSynchronize());
#if TIMECOUNTING  
  timer->Done("Calculate nodes array for one-way unzipped edges");
#endif
  uint64_t result = 0;

  if (device_count == 1) {
    uint64_t* dev_results;
    CUCHECK(hipMalloc(&dev_results,
          NUM_BLOCKS * NUM_THREADS * sizeof(uint64_t)));
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(CalculateTriangles), hipFuncCachePreferL1);
    hipProfilerStart();
    CalculateTriangles<<<NUM_BLOCKS, NUM_THREADS>>>(
        m, dev_edges, dev_nodes, dev_results);
    CUCHECK(hipDeviceSynchronize());
    hipProfilerStop();
#if TIMECOUNTING    
    timer->Done("Calculate triangles");
#endif
    result = SumResults(NUM_BLOCKS * NUM_THREADS, dev_results);
#if TIMECOUNTING    
    CUCHECK(hipFree(dev_results));
#endif    
    timer->Done("Reduce");
  } else {
    result = MultiGPUCalculateTriangles(
        n, m, dev_edges, dev_nodes, device_count);
#if TIMECOUNTING        
    timer->Done("Calculate triangles on multi GPU");
#endif    
  }

  CUCHECK(hipFree(dev_edges));
  CUCHECK(hipFree(dev_nodes));

  delete timer;

  return result;
}

void PreInitGpuContext(int device) {
  CUCHECK(hipSetDevice(device));
  CUCHECK(hipFree(NULL));
}
